//Вариант 20.	Задать 1024 числа и выдать чётное/ нечётное

#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void kernel(int n, int* array_odd, int* array_even) 
{
    long long i = blockDim.x * blockIdx.x + threadIdx.x;
    long long offset = blockDim.x * gridDim.x;
    int c_even = 0;
    int c_odd = 0;
    for (; i < n; i += offset)
    {
        if (i % 2 == 0)
        {
            array_odd[c_odd] = i;
            c_odd++;
        }
        else
        {
            array_even[c_even] = i;
            c_even++;
        }
    }
}


int main() {
    float time;
    int n = 1024;
    int blocks = 1, threads = 2;

    int* c_array_odd = (int*)malloc(n/2 * sizeof(int));
    int* c_array_even = (int*)malloc(n/2 * sizeof(int));
    int* cu_array_odd;
    int* cu_array_even;
    hipMalloc(&cu_array_odd, sizeof(int) * n/2);
    hipMalloc(&cu_array_even, sizeof(int) * n/2);
    hipMemcpy(cu_array_odd, c_array_odd, sizeof(int) * n/2, hipMemcpyHostToDevice); // from CPU to GPU
    hipMemcpy(cu_array_even, c_array_even, sizeof(int) * n/2, hipMemcpyHostToDevice); // from CPU to GPU

    hipEvent_t start, end;
    hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

    kernel<<<blocks,threads>>>(n, cu_array_even, cu_array_odd);
    hipEventRecord(end);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);
 
    printf("kernel = <<<%d, %d>>>, time = %f\n", blocks, threads, time);

    hipMemcpy(c_array_odd, cu_array_odd, sizeof(int) * n/2, hipMemcpyDeviceToHost); // From GPU to CPU
    hipMemcpy(c_array_even, cu_array_even, sizeof(int) * n/2, hipMemcpyDeviceToHost); // From GPU to CPU
    hipFree(cu_array_odd);
    hipFree(cu_array_even);

    printf("Even numbers:\n");
    for (int i = 0; i < n/2; i++) {
		printf("%d ", c_array_even[i]);
	}

    printf("\n Odd numbers:\n");
    for (int i = 0; i < n/2; i++) {
		printf("%d ", c_array_odd[i]);
	}
    printf("\n");


    free(c_array_odd);
    free(c_array_even);

    system ( "PAUSE" );
    return 0;
}